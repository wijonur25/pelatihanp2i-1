#include <stdio.h>
#include <hip/hip_runtime.h>
#define N 12
// tugas 1: alokasi memori dan transfer dari device ke host

__global__ void
kern(int *A)
{
        int idx = blockDim.x * blockIdx.x + threadIdx.x;
        A[idx] = idx;
}

/**
 * Host main routine
 */
int   main(void)
{
        // alokasikan memori, dan salin nilainya
        int *A = (int *) malloc(N*sizeof(int)); //alokasi memori di host
        int *d_A;
        hipMalloc(&d_A,N*sizeof(int)); //alokasi memori di device
        hipMemcpy(d_A,A,N*sizeof(int),hipMemcpyHostToDevice);

        dim3 grid,block;
        block.x = 4;
        grid.x = 12/block.x;
        kern<<<grid,block>>>(d_A);
        hipMemcpy(A,d_A,N*sizeof(int),hipMemcpyDeviceToHost);
        // copy result
        for(int i = 0;i < N;i++)
                printf("A[%d] = %d\n",i,A[i]);
        free(A);
        hipFree(d_A);
        return 0;
}
